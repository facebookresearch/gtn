#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <algorithm>
#include <cassert>
#include <numeric>
#include <vector>
#include <tuple>
#include <iostream>

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>

#include "parallel_compose.h"
#include "prefix_scan.h"

namespace gtn {
namespace detail {
namespace dataparallel {

namespace {

struct GraphDataParallelGPU {
  size_t numNodes;
  size_t numArcs;

  // True if a node is accept or start, false otherwise
  int* accept;
  int* start;

  // One value per node - i-th value corresponds to i-th node
  // Last element is the total number of arcs, so that
  // each element and its neighbor forms a range
  int* inArcOffset;
  int* outArcOffset;

  // One value per arc
  int* inArcs;
  int* outArcs;

  // One value per arc
  // i-th value corresponds to i-th arc
  int* ilabels;
  int* olabels;
  int* srcNodes;
  int* dstNodes;
  float* weights;
};

struct nodeAndArcPairGPU {
  int2 nodePair;
  int2 arcPair;
  int2 checkEpsilonArcPair;
  bool checkArcPair;
  bool isValid;
};

inline int div_up(int x, int y) {
  return (x + y - 1) / y;
}

__device__ __host__
inline int TwoDToOneDIndex(int n1, int n2, int n1Extent) {
  assert(n1 < n1Extent);
  return n1 + n2 * n1Extent;
}

inline std::pair<int, int> OneDToTwoDIndex(int n, int n1Extent) {
  assert(n1Extent > 0);
  const int n2 = n / n1Extent;
  const int n1 = n % n1Extent;
  return std::make_pair(n1, n2);
}

bool checkAnyTrue(const std::vector<int>& flags) {
  // Potentially wasteful - but GPU friendly
  return std::accumulate(flags.begin(), flags.end(), 0) > 0 ? true : false;
}

bool checkAnyTrueGPU(const int* flags, int numFlags) {
  thrust::device_ptr<const int> tPtr(flags);
  const int sum = thrust::reduce(tPtr, tPtr + numFlags, int(0));

  return (sum > 0);
}

std::tuple<int*, size_t, int> prefixSumScanGPU(const int* input, size_t numElts, bool appendSum) {
  assert(numElts > 0);
  const size_t scanNumElts = appendSum ? numElts + 1 : numElts;

  int *output;
  hipMalloc((void **)(&(output)), sizeof(int) * scanNumElts);
  hipMemcpy((void *)(output), (void *)(input), sizeof(int) * numElts, hipMemcpyDeviceToDevice);

  thrust::device_ptr<int> tPtr(output);
  thrust::exclusive_scan(tPtr, tPtr + numElts, tPtr);

  int lastElementInput;
  int lastElementOutput;
  hipMemcpy((void *)(&lastElementInput), (void *)(&(input[numElts-1])), sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)(&lastElementOutput), (void *)(&(output[numElts-1])), sizeof(int), hipMemcpyDeviceToHost);
  const int sum = lastElementInput + lastElementOutput;

  if (appendSum) {
    hipMemcpy((void *)(&(output[scanNumElts-1])), (void *)(&sum), sizeof(int), hipMemcpyHostToDevice);
  }

  return std::make_tuple(output, scanNumElts, sum);
}


// Map thread id to corresponding node and arc pair
// Also map thread id to two flags checkEpsilonArcPair.first,
// checkEpsilonArcPair.second When checkEpsilonArcPair.first is set,
// corresponding tid will check for arcs with epsilon arcs in the node from
// first graph Same logic happens for checkEpsilonArcPair.second Search to find
// which node pair this tid will fall into Linear search for now
// (arcCrossProductOffset is sorted by definition)
__device__
nodeAndArcPairGPU computeNodeAndArcPair(
    int tid,
    size_t numArcCrossProductOffset,
    const int* arcCrossProductOffset,
    const int* toExploreNumArcsFirst,
    const int* toExploreNumArcsSecond,
    const int* toExploreNodePairFirst,
    const int* toExploreNodePairSecond) {

  nodeAndArcPairGPU result;
  result.checkArcPair = false;
  result.checkEpsilonArcPair = make_int2(false, false);
  result.isValid = false;

  // There should be at least two values to form a range
  assert(numArcCrossProductOffset >= 2);

  for (size_t i = 0; i < numArcCrossProductOffset - 1; ++i) {
    const int lVal = arcCrossProductOffset[i];
    const int rVal = arcCrossProductOffset[i + 1];

    if ((lVal <= tid) && (tid < rVal)) {
      result.isValid = true;
      result.nodePair = make_int2(
          toExploreNodePairFirst[i], toExploreNodePairSecond[i]);

      // The range of idx is from
      // [0, toExploreNumArcsFirst[i] * toExploreNumArcsSecond[i])
      const int idx = tid - lVal;
      const int numArcs = rVal - lVal;

      assert(idx >= 0);
      assert(idx < numArcs);
      assert(numArcs > 0);

      const int arcProd =
          toExploreNumArcsFirst[i] * toExploreNumArcsSecond[i];

      if (numArcs == arcProd) {
        result.checkArcPair = true;

        // We map the tids to 2D grid where the
        // x-axis is toExploreNumArcsFirst[i] (row)
        // y-axis is toExploreNumArcsSecond[i] (column)
	assert(toExploreNumArcsFirst[i] > 0);
        result.arcPair = make_int2(
	    idx % toExploreNumArcsFirst[i],
	    idx / toExploreNumArcsFirst[i]);

        // Pick the tids from the first row since we need only one
        // tid per arc of the node from the first graph to check for
        // epsilon
        if (idx < toExploreNumArcsFirst[i]) {
          result.checkEpsilonArcPair.x = true;
        }

        // Pick the tids from the first column since we need only one
        // tid per arc of the node from the first graph to check for
        // epsilon
        if ((idx % toExploreNumArcsFirst[i]) == 0) {
          result.checkEpsilonArcPair.y = true;
        }
      } else if ((arcProd == 0) && (numArcs == toExploreNumArcsFirst[i])) {
        // TODO: Likely not the brightest idea to use -1 as sentinel
        result.arcPair = make_int2(idx, -1);
        result.checkEpsilonArcPair.x = true;
      } else if ((arcProd == 0) && (numArcs == toExploreNumArcsSecond[i])) {
        // TODO: Likely not the brightest idea to use -1 as sentinel
        result.arcPair = make_int2(-1, idx);
        result.checkEpsilonArcPair.y = true;
      }

      break;
    }
  }

  return result;
}

// Takes a pair of nodes, where each member of pair comes from a different
// graph and calculate a vector of number of arcs in the cross product of
// arcs outgoing from each pair.
// This should be a kernel call
std::tuple<std::vector<int>, std::pair<std::vector<int>, std::vector<int>>>
calculateArcCrossProductOffset(
    const std::pair<std::vector<int>, std::vector<int>>& toExploreNodePair,
    const GraphDataParallel& graphDP1,
    const GraphDataParallel& graphDP2,
    bool inOrOutArc) {
  assert(toExploreNodePair.first.size() == toExploreNodePair.second.size());

  std::pair<std::vector<int>, std::vector<int>> toExploreNumArcs;
  toExploreNumArcs.first.resize(toExploreNodePair.first.size());
  toExploreNumArcs.second.resize(toExploreNodePair.first.size());

  std::vector<int> arcCrossProductOffset(toExploreNodePair.first.size());

  // No dependence between iterations
  for (size_t i = 0; i < toExploreNodePair.first.size(); ++i) {
    int node = (toExploreNodePair.first)[i];
    // Special case if it is the last node. Then the offset becomes
    // the number of arcs
    const int inArcOffsetGraph1 = ((node + 1) == graphDP1.inArcOffset.size())
        ? graphDP1.inArcs.size()
        : graphDP1.inArcOffset[node + 1];
    const int outArcOffsetGraph1 = ((node + 1) == graphDP1.outArcOffset.size())
        ? graphDP1.outArcs.size()
        : graphDP1.outArcOffset[node + 1];

    const int numArcsFirst = inOrOutArc
        ? inArcOffsetGraph1 - graphDP1.inArcOffset[node]
        : outArcOffsetGraph1 - graphDP1.outArcOffset[node];

    node = (toExploreNodePair.second)[i];
    // Special case if it is the last node. Then the offset becomes
    // the number of arcs
    const int inArcOffsetGraph2 = ((node + 1) == graphDP2.inArcOffset.size())
        ? graphDP2.inArcs.size()
        : graphDP2.inArcOffset[node + 1];
    const int outArcOffsetGraph2 = ((node + 1) == graphDP2.outArcOffset.size())
        ? graphDP2.outArcs.size()
        : graphDP2.outArcOffset[node + 1];

    const int numArcsSecond = inOrOutArc
        ? inArcOffsetGraph2 - graphDP2.inArcOffset[node]
        : outArcOffsetGraph2 - graphDP2.outArcOffset[node];

    (toExploreNumArcs.first)[i] = numArcsFirst;
    (toExploreNumArcs.second)[i] = numArcsSecond;

    // Even when numArcsFirst or numArcsSecond is 0 we have to consider
    // the case when the other graph has arcs with epsilon label
    if (numArcsFirst != 0 && numArcsSecond != 0) {
      arcCrossProductOffset[i] = numArcsFirst * numArcsSecond;
    } else if (numArcsFirst != 0 && numArcsSecond == 0) {
      arcCrossProductOffset[i] = numArcsFirst;
    } else if (numArcsFirst == 0 && numArcsSecond != 0) {
      arcCrossProductOffset[i] = numArcsSecond;
    } else {
      arcCrossProductOffset[i] = 0;
    }
  }

  return std::make_tuple(arcCrossProductOffset, toExploreNumArcs);
}

__global__
void calculateArcCrossProductOffsetKernel(
      const GraphDataParallelGPU graphDP1GPU,
      const GraphDataParallelGPU graphDP2GPU,
      const int* toExploreNodePairFirstGPU,
      const int* toExploreNodePairSecondGPU,
      int* toExploreNumArcsFirstGPU,
      int* toExploreNumArcsSecondGPU,
      int* arcCrossProductOffsetGPU,
      size_t numToExploreNodePair,
      bool inOrOutArc) {
  const int gTid = blockIdx.x * blockDim.x + threadIdx.x;

  if (gTid < numToExploreNodePair) {
    int node = toExploreNodePairFirstGPU[gTid];
    // Special case if it is the last node. Then the offset becomes
    // the number of arcs
    const int inArcOffsetGraph1 = ((node + 1) == graphDP1GPU.numNodes)
        ? graphDP1GPU.numArcs
        : graphDP1GPU.inArcOffset[node + 1];
    const int outArcOffsetGraph1 = ((node + 1) == graphDP1GPU.numNodes)
        ? graphDP1GPU.numArcs
        : graphDP1GPU.outArcOffset[node + 1];

    const int numArcsFirst = inOrOutArc
        ? inArcOffsetGraph1 - graphDP1GPU.inArcOffset[node]
        : outArcOffsetGraph1 - graphDP1GPU.outArcOffset[node];

    node = toExploreNodePairSecondGPU[gTid];
    // Special case if it is the last node. Then the offset becomes
    // the number of arcs
    const int inArcOffsetGraph2 = ((node + 1) == graphDP2GPU.numNodes)
        ? graphDP2GPU.numArcs
        : graphDP2GPU.inArcOffset[node + 1];
    const int outArcOffsetGraph2 = ((node + 1) == graphDP2GPU.numNodes)
        ? graphDP2GPU.numArcs
        : graphDP2GPU.outArcOffset[node + 1];

    const int numArcsSecond = inOrOutArc
        ? inArcOffsetGraph2 - graphDP2GPU.inArcOffset[node]
        : outArcOffsetGraph2 - graphDP2GPU.outArcOffset[node];

    toExploreNumArcsFirstGPU[gTid] = numArcsFirst;
    toExploreNumArcsSecondGPU[gTid] = numArcsSecond;

    // Even when numArcsFirst or numArcsSecond is 0 we have to consider
    // the case when the other graph has arcs with epsilon label
    if (numArcsFirst != 0 && numArcsSecond != 0) {
      arcCrossProductOffsetGPU[gTid] = numArcsFirst * numArcsSecond;
    } else if (numArcsFirst != 0 && numArcsSecond == 0) {
      arcCrossProductOffsetGPU[gTid] = numArcsFirst;
    } else if (numArcsFirst == 0 && numArcsSecond != 0) {
      arcCrossProductOffsetGPU[gTid] = numArcsSecond;
    } else {
      arcCrossProductOffsetGPU[gTid] = 0;
    }
  }
}

// Takes a pair of nodes, where each member of pair comes from a different
// graph and calculate a vector of number of arcs in the cross product of
// arcs outgoing from each pair.
// This should be a kernel call
std::tuple<int*, int*, int*>
calculateArcCrossProductOffsetGPU(
    const int* toExploreNodePairFirstGPU,
    const int* toExploreNodePairSecondGPU,
    size_t numToExploreNodePair,
    const GraphDataParallelGPU graphDP1GPU,
    const GraphDataParallelGPU graphDP2GPU,
    bool inOrOutArc) {

  int* toExploreNumArcsFirstGPU;
  int* toExploreNumArcsSecondGPU;
  int* arcCrossProductOffsetGPU;
  hipMalloc((void **)(&(toExploreNumArcsFirstGPU)), sizeof(int) * numToExploreNodePair);
  hipMalloc((void **)(&(toExploreNumArcsSecondGPU)), sizeof(int) * numToExploreNodePair);
  hipMalloc((void **)(&(arcCrossProductOffsetGPU)), sizeof(int) * numToExploreNodePair);

  const int NT = 128;
  const int gridSize = div_up(numToExploreNodePair, NT);

  calculateArcCrossProductOffsetKernel<<<gridSize, NT, 0, 0>>>(
      graphDP1GPU, graphDP2GPU, toExploreNodePairFirstGPU, toExploreNodePairSecondGPU,
      toExploreNumArcsFirstGPU, toExploreNumArcsSecondGPU, arcCrossProductOffsetGPU,
      numToExploreNodePair, inOrOutArc);

  return std::make_tuple(arcCrossProductOffsetGPU, toExploreNumArcsFirstGPU, toExploreNumArcsSecondGPU);
}

// This function needs to be thread safe since multiple threads can
// can call it and they will overlap on curIdx and dstIdx
__device__
void calculateNumArcsAndNodesToExplore(
    int curIdx,
    int dstIdx,
    const int* reachable,
    int* newNodes,
    int* toExplore,
    int* numOutArcs,
    int* numInArcs) {
  if (reachable[dstIdx]) {
    // Atomic test and set for newNodes
    /*
    int oldVal = newNodes[dstIdx];
    if (!newNodes[dstIdx]) {
      newNodes[dstIdx] = true;
    }*/
    int oldVal = atomicCAS(&(newNodes[dstIdx]), false, true);
    if (!oldVal) {
      toExplore[dstIdx] = true;
    }

    // These are atomic increments
    // numOutArcs[curIdx]++;
    // numInArcs[dstIdx]++;
    atomicAdd(&(numOutArcs[curIdx]), 1);
    atomicAdd(&(numInArcs[dstIdx]), 1);

    // printf("cidx %d didx %d\n", curIdx, dstIdx);
    // printf("no %d ni %d\n", numOutArcs[curIdx], numInArcs[dstIdx]);
  }
}

// This function needs to be thread safe since multiple threads can
// can call it
__device__
void generateCombinedGraphNodesAndArcs(
    int dstIdx,
    int curIdx,
    const int2& arcPair,
    const int2& dstNodeStartAndAccept,
    const int* reachable,
    const int* newNodesOffset,
    int* newNodesVisited,
    int* toExplore,
    int* gradInfoFirst,
    int* gradInfoSecond,
    GraphDataParallelGPU& newGraphDP,
    int ilabel,
    int olabel,
    float weight) {
  if (reachable[dstIdx]) {
    // Atomic test and set for newNodesVisited
    /*
    int oldVal = newNodesVisited[dstIdx];
    if (!newNodesVisited[dstIdx]) {
      newNodesVisited[dstIdx] = true;
    }*/

    int oldVal = atomicCAS(&(newNodesVisited[dstIdx]), false, true);
    if (!oldVal) {
      toExplore[dstIdx] = true;
    }

    // Set accept and start nodes
    // I think I only need it for dst nodes and src nodes
    // Note: Multiple threads can have the same dstIdx and write to the same
    //       location and collide. This _should_ be fine since they are going
    //       to write the same value
    newGraphDP.start[newNodesOffset[dstIdx]] = dstNodeStartAndAccept.x;
    newGraphDP.accept[newNodesOffset[dstIdx]] = dstNodeStartAndAccept.y;

    // Both of these increments are atomic
    // int inArcIdx = newGraphDP.inArcOffset[newNodesOffset[dstIdx]]++;
    // int outArcIdx = newGraphDP.outArcOffset[newNodesOffset[curIdx]]++;

    int inArcIdx = atomicAdd(&(newGraphDP.inArcOffset[newNodesOffset[dstIdx]]), 1);
    int outArcIdx = atomicAdd(&(newGraphDP.outArcOffset[newNodesOffset[curIdx]]), 1);

    // printf("dstIdx %d curIdx %d\n", dstIdx, curIdx);
    // printf("inArcIdx %d outArcIdx %d\n", inArcIdx, outArcIdx);

    // outArcIdx is also the arc identifier
    newGraphDP.outArcs[outArcIdx] = outArcIdx;
    newGraphDP.inArcs[inArcIdx] = outArcIdx;

    // Fill in everything else for this arc
    newGraphDP.ilabels[outArcIdx] = ilabel;
    newGraphDP.olabels[outArcIdx] = olabel;
    newGraphDP.srcNodes[outArcIdx] = newNodesOffset[curIdx];
    newGraphDP.dstNodes[outArcIdx] = newNodesOffset[dstIdx];
    newGraphDP.weights[outArcIdx] = weight;

    // printf("ilabels %d olabels %d srcNodes %d dstNodes %d weights %f\n",
           // newGraphDP.ilabels[outArcIdx], newGraphDP.olabels[outArcIdx],
	   // newGraphDP.srcNodes[outArcIdx], newGraphDP.dstNodes[outArcIdx],
	   // newGraphDP.weights[outArcIdx]);

    gradInfoFirst[outArcIdx] = arcPair.x;
    gradInfoSecond[outArcIdx] = arcPair.y;
  }
}

// Convert bool array two pairs for true flags
std::pair<std::vector<int>, std::vector<int>> convertToNodePair(
    const std::vector<int>& flags,
    int extent) {
  std::pair<std::vector<int>, std::vector<int>> toExploreNodePair;
  for (size_t i = 0; i < flags.size(); ++i) {
    if (flags[i] == true) {
      std::pair<int, int> node = OneDToTwoDIndex(i, extent);
      toExploreNodePair.first.push_back(node.first);
      toExploreNodePair.second.push_back(node.second);
    }
  }

  return toExploreNodePair;
}

// Takes a bool array with flags set for nodes to pick and returns
// an array with indices that were set as true
std::vector<int> convertToNodes(const std::vector<int>& flags) {
  std::vector<int> nodes;

  for (size_t i = 0; i < flags.size(); ++i) {
    if (flags[i]) {
      nodes.push_back(i);
    }
  }

  return nodes;
}

__device__
int2 getStartAndAccept(
    const GraphDataParallelGPU& graphDP1,
    const GraphDataParallelGPU& graphDP2,
    const int2& dstNodePair) {

  int2 dstNodeStartAndAccept = make_int2(
      graphDP1.start[dstNodePair.x] && graphDP2.start[dstNodePair.y],
      graphDP1.accept[dstNodePair.x] &&
          graphDP2.accept[dstNodePair.y]);

  return dstNodeStartAndAccept;
}

GraphDataParallelGPU copyToGPU(const GraphDataParallel& graphDP) {
  GraphDataParallelGPU graphDPGPU;

  graphDPGPU.numNodes = graphDP.inArcOffset.size();
  graphDPGPU.numArcs = graphDP.inArcs.size();

  assert(graphDP.accept.size() == graphDPGPU.numNodes);
  assert(graphDP.start.size() == graphDPGPU.numNodes);
  assert(graphDP.inArcOffset.size() == graphDPGPU.numNodes);
  assert(graphDP.outArcOffset.size() == graphDPGPU.numNodes);

  assert(graphDP.inArcs.size() == graphDPGPU.numArcs);
  assert(graphDP.outArcs.size() == graphDPGPU.numArcs);
  assert(graphDP.ilabels.size() == graphDPGPU.numArcs);
  assert(graphDP.olabels.size() == graphDPGPU.numArcs);
  assert(graphDP.srcNodes.size() == graphDPGPU.numArcs);
  assert(graphDP.dstNodes.size() == graphDPGPU.numArcs);
  assert(graphDP.weights.size() == graphDPGPU.numArcs);

  // Allocate memory
  hipMalloc((void **)(&(graphDPGPU.accept)), sizeof(int) * graphDPGPU.numNodes);

  hipMalloc((void **)(&(graphDPGPU.start)), sizeof(int) * graphDPGPU.numNodes);

  hipMalloc((void **)(&(graphDPGPU.inArcOffset)), sizeof(int) * graphDPGPU.numNodes);
  hipMalloc((void **)(&(graphDPGPU.outArcOffset)), sizeof(int) * graphDPGPU.numNodes);

  hipMalloc((void **)(&(graphDPGPU.inArcs)), sizeof(int) * graphDPGPU.numArcs);
  hipMalloc((void **)(&(graphDPGPU.outArcs)), sizeof(int) * graphDPGPU.numArcs);

  hipMalloc((void **)(&(graphDPGPU.ilabels)), sizeof(int) * graphDPGPU.numArcs);
  hipMalloc((void **)(&(graphDPGPU.olabels)), sizeof(int) * graphDPGPU.numArcs);
  hipMalloc((void **)(&(graphDPGPU.srcNodes)), sizeof(int) * graphDPGPU.numArcs);
  hipMalloc((void **)(&(graphDPGPU.dstNodes)), sizeof(int) * graphDPGPU.numArcs);
  hipMalloc((void **)(&(graphDPGPU.weights)), sizeof(float) * graphDPGPU.numArcs);

  // Copy
  hipMemcpy((void *)(graphDPGPU.accept), (void *)(graphDP.accept.data()), sizeof(int) * graphDPGPU.numNodes, hipMemcpyHostToDevice);
  hipMemcpy((void *)(graphDPGPU.start), (void *)(graphDP.start.data()), sizeof(int) * graphDPGPU.numNodes, hipMemcpyHostToDevice);

  hipMemcpy((void *)(graphDPGPU.inArcOffset), (void *)(graphDP.inArcOffset.data()), sizeof(int) * graphDPGPU.numNodes, hipMemcpyHostToDevice);
  hipMemcpy((void *)(graphDPGPU.outArcOffset), (void *)(graphDP.outArcOffset.data()), sizeof(int) * graphDPGPU.numNodes, hipMemcpyHostToDevice);

  hipMemcpy((void *)(graphDPGPU.inArcs), (void *)(graphDP.inArcs.data()), sizeof(int) * graphDPGPU.numArcs, hipMemcpyHostToDevice);
  hipMemcpy((void *)(graphDPGPU.outArcs), (void *)(graphDP.outArcs.data()), sizeof(int) * graphDPGPU.numArcs, hipMemcpyHostToDevice);

  hipMemcpy((void *)(graphDPGPU.ilabels), (void *)(graphDP.ilabels.data()), sizeof(int) * graphDPGPU.numArcs, hipMemcpyHostToDevice);
  hipMemcpy((void *)(graphDPGPU.olabels), (void *)(graphDP.olabels.data()), sizeof(int) * graphDPGPU.numArcs, hipMemcpyHostToDevice);
  hipMemcpy((void *)(graphDPGPU.srcNodes), (void *)(graphDP.srcNodes.data()), sizeof(int) * graphDPGPU.numArcs, hipMemcpyHostToDevice);
  hipMemcpy((void *)(graphDPGPU.dstNodes), (void *)(graphDP.dstNodes.data()), sizeof(int) * graphDPGPU.numArcs, hipMemcpyHostToDevice);
  hipMemcpy((void *)(graphDPGPU.weights), (void *)(graphDP.weights.data()), sizeof(float) * graphDPGPU.numArcs, hipMemcpyHostToDevice);

  return graphDPGPU;
}

__global__ 
void findReachableKernel(
      const GraphDataParallelGPU graphDP1GPU,
      const GraphDataParallelGPU graphDP2GPU,
      const int* arcCrossProductOffsetGPU,
      const int* toExploreNumArcsFirstGPU,
      const int* toExploreNumArcsSecondGPU,
      const int* toExploreNodePairFirstGPU,
      const int* toExploreNodePairSecondGPU,
      int numNodesFirst,
      int totalArcs,
      size_t numArcCrossProductOffset,
      int* toExploreGPU,
      int* reachableGPU,
      int* epsilonMatchedGPU
      ) {
  const int gTid = blockIdx.x * blockDim.x + threadIdx.x;

  if (gTid < totalArcs) {
    nodeAndArcPairGPU result = 
      computeNodeAndArcPair(
        gTid, numArcCrossProductOffset, arcCrossProductOffsetGPU,
        toExploreNumArcsFirstGPU, toExploreNumArcsSecondGPU,
        toExploreNodePairFirstGPU, toExploreNodePairSecondGPU);

    // printf("tid = %d, valid = %d\n", gTid, result.isValid);
    // Does this node pair match?
    if (result.isValid) {

      int inArcOffset = graphDP1GPU.inArcOffset[result.nodePair.x];
      const int firstArcIdx = graphDP1GPU.inArcs[inArcOffset + result.arcPair.x];

      inArcOffset = graphDP2GPU.inArcOffset[result.nodePair.y];
      const int secondArcIdx = graphDP2GPU.inArcs[inArcOffset + result.arcPair.y];

      // printf("tid = %d, cp = %d\n", gTid, result.checkArcPair);

      if (result.checkArcPair &&
          (graphDP1GPU.olabels[firstArcIdx] == graphDP2GPU.ilabels[secondArcIdx])) {
        const int idx = TwoDToOneDIndex(
            graphDP1GPU.srcNodes[firstArcIdx],
            graphDP2GPU.srcNodes[secondArcIdx],
            numNodesFirst);

	// printf("tid = %d, idx = %d\n", gTid, idx);

        if (graphDP1GPU.olabels[firstArcIdx] == epsilon) {
          epsilonMatchedGPU[idx] = true;
        }

        // idx may not be unique amongst all threads.
        /*
        int oldVal = reachableGPU[idx];
        if (!reachableGPU[idx]) {
          reachableGPU[idx] = true;
        }*/
        int oldVal = atomicCAS(&(reachableGPU[idx]), false, true);
        if (!oldVal) {
          toExploreGPU[idx] = true;
        }
	// printf("r %d t %d \n", reachableGPU[idx], toExploreGPU[idx]);
      }

      // Only valid for arcs incoming to node from first graph
      if (result.checkEpsilonArcPair.x &&
          (graphDP1GPU.olabels[firstArcIdx] == epsilon)) {
        const int idx = TwoDToOneDIndex(
            graphDP1GPU.srcNodes[firstArcIdx], result.nodePair.y, numNodesFirst);
        /*
        int oldVal = reachableGPU[idx];
        if (!reachableGPU[idx]) {
          reachableGPU[idx] = true;
        }*/
        int oldVal = atomicCAS(&(reachableGPU[idx]), false, true);
        if (!oldVal) {
          toExploreGPU[idx] = true;
        }
      }

      // Only valid for arcs incoming to node from second graph
      if (result.checkEpsilonArcPair.y &&
          (graphDP2GPU.ilabels[secondArcIdx] == epsilon)) {
        const int idx = TwoDToOneDIndex(
            result.nodePair.x, graphDP2GPU.srcNodes[secondArcIdx], numNodesFirst);
        /*
        int oldVal = reachableGPU[idx];
        if (!reachableGPU[idx]) {
          reachableGPU[idx] = true;
        }*/
        int oldVal = atomicCAS(&(reachableGPU[idx]), false, true);
        if (!oldVal) {
          toExploreGPU[idx] = true;
        }
      }
    }
  }
}

__global__ 
void computeValidNodeAndArcKernel(
      const GraphDataParallelGPU graphDP1GPU,
      const GraphDataParallelGPU graphDP2GPU,
      const int* arcCrossProductOffsetGPU,
      const int* toExploreNumArcsFirstGPU,
      const int* toExploreNumArcsSecondGPU,
      const int* toExploreNodePairFirstGPU,
      const int* toExploreNodePairSecondGPU,
      const int* reachableGPU,
      const int* epsilonMatchedGPU,
      int numNodesFirst,
      int totalArcs,
      size_t numArcCrossProductOffset,
      int* toExploreGPU,
      int* newNodesGPU,
      int* numInArcsGPU,
      int* numOutArcsGPU
      ) {
  const int gTid = blockIdx.x * blockDim.x + threadIdx.x;

  if (gTid < totalArcs) {
    // Map tid to corresponding node and arc pair
    // Search to find which node pair this tid will fall into
    nodeAndArcPairGPU result = 
      computeNodeAndArcPair(
        gTid, numArcCrossProductOffset, arcCrossProductOffsetGPU,
        toExploreNumArcsFirstGPU, toExploreNumArcsSecondGPU,
        toExploreNodePairFirstGPU, toExploreNodePairSecondGPU);

    if (result.isValid) {
      int outArcOffset = graphDP1GPU.outArcOffset[result.nodePair.x];
      const int firstArcIdx = graphDP1GPU.outArcs[outArcOffset + result.arcPair.x];

      outArcOffset = graphDP2GPU.outArcOffset[result.nodePair.y];
      const int secondArcIdx =
          graphDP2GPU.outArcs[outArcOffset + result.arcPair.y];

      const bool epsilonMatch = epsilonMatchedGPU[TwoDToOneDIndex(
          result.nodePair.x, result.nodePair.y, numNodesFirst)];

      // Does this node pair match?
      // Skip epsilon matches
      if (result.checkArcPair &&
          (graphDP1GPU.olabels[firstArcIdx] == graphDP2GPU.ilabels[secondArcIdx])) {
        const int dstIdx = TwoDToOneDIndex(
            graphDP1GPU.dstNodes[firstArcIdx],
            graphDP2GPU.dstNodes[secondArcIdx],
            numNodesFirst);
        const int curIdx =
            TwoDToOneDIndex(result.nodePair.x, result.nodePair.y, numNodesFirst);

        // printf("krnl 1a dst %d cur %d\n", dstIdx, curIdx);

        // We track if any two arcs outgoing from this node pair match
        // on epsilon. We record if they do.
        if (graphDP1GPU.olabels[firstArcIdx] != epsilon) {
          calculateNumArcsAndNodesToExplore(
              curIdx,
              dstIdx,
              reachableGPU,
              newNodesGPU,
              toExploreGPU,
              numOutArcsGPU,
              numInArcsGPU);
        }
      }

      if (result.checkEpsilonArcPair.x &&
          (!epsilonMatch || graphDP2GPU.accept[result.nodePair.y] ||
           !graphDP1GPU.accept[result.nodePair.x]) &&
          (graphDP1GPU.olabels[firstArcIdx] == epsilon)) {
        const int dstIdx = TwoDToOneDIndex(
            graphDP1GPU.dstNodes[firstArcIdx], result.nodePair.y, numNodesFirst);
        const int curIdx =
            TwoDToOneDIndex(result.nodePair.x, result.nodePair.y, numNodesFirst);

        // printf("krnl 1b dst %d cur %d\n", dstIdx, curIdx);

        calculateNumArcsAndNodesToExplore(
            curIdx,
            dstIdx,
            reachableGPU,
            newNodesGPU,
            toExploreGPU,
            numOutArcsGPU,
            numInArcsGPU);
      }

      if (result.checkEpsilonArcPair.y &&
          (!epsilonMatch || graphDP1GPU.accept[result.nodePair.x]) &&
          (graphDP2GPU.ilabels[secondArcIdx] == epsilon)) {
        const int dstIdx = TwoDToOneDIndex(
            result.nodePair.x, graphDP2GPU.dstNodes[secondArcIdx], numNodesFirst);
        const int curIdx =
            TwoDToOneDIndex(result.nodePair.x, result.nodePair.y, numNodesFirst);

        // printf("krnl 1c dst %d cur %d\n", dstIdx, curIdx);

        calculateNumArcsAndNodesToExplore(
            curIdx,
            dstIdx,
            reachableGPU,
            newNodesGPU,
            toExploreGPU,
            numOutArcsGPU,
            numInArcsGPU);
      }
    }
  }
}

__global__ 
void generateNodeAndArcKernel(
      const GraphDataParallelGPU graphDP1GPU,
      const GraphDataParallelGPU graphDP2GPU,
      const int* arcCrossProductOffsetGPU,
      const int* toExploreNumArcsFirstGPU,
      const int* toExploreNumArcsSecondGPU,
      const int* toExploreNodePairFirstGPU,
      const int* toExploreNodePairSecondGPU,
      const int* reachableGPU,
      const int* epsilonMatchedGPU,
      int numNodesFirst,
      int totalArcs,
      size_t numArcCrossProductOffset,
      GraphDataParallelGPU newGraphDPGPU,
      int* toExploreGPU,
      int* gradInfoFirstGPU,
      int* gradInfoSecondGPU,
      int* newNodesOffsetGPU,
      int* newNodesVisitedGPU
      ) {
  const int gTid = blockIdx.x * blockDim.x + threadIdx.x;

  if (gTid < totalArcs) {
    // Map tid to corresponding node and arc pair
    // Search to find which node pair this tid will fall into
    nodeAndArcPairGPU result = 
      computeNodeAndArcPair(
        gTid, numArcCrossProductOffset, arcCrossProductOffsetGPU,
        toExploreNumArcsFirstGPU, toExploreNumArcsSecondGPU,
        toExploreNodePairFirstGPU, toExploreNodePairSecondGPU);

    if (result.isValid) {
      int outArcOffset = graphDP1GPU.outArcOffset[result.nodePair.x];
      const int firstArcIdx = graphDP1GPU.outArcs[outArcOffset + result.arcPair.x];

      outArcOffset = graphDP2GPU.outArcOffset[result.nodePair.y];
      const int secondArcIdx =
          graphDP2GPU.outArcs[outArcOffset + result.arcPair.y];

      const bool epsilonMatch = epsilonMatchedGPU[TwoDToOneDIndex(
          result.nodePair.x, result.nodePair.y, numNodesFirst)];

      // Does this node pair match?
      if (result.checkArcPair &&
          (graphDP1GPU.olabels[firstArcIdx] == graphDP2GPU.ilabels[secondArcIdx])) {
        int2 dstNodePair = make_int2(
            graphDP1GPU.dstNodes[firstArcIdx], graphDP2GPU.dstNodes[secondArcIdx]);

        const int dstIdx = TwoDToOneDIndex(
            dstNodePair.x, dstNodePair.y, numNodesFirst);
        const int curIdx = TwoDToOneDIndex(
            result.nodePair.x, result.nodePair.y, numNodesFirst);

	// printf("krn2a dstIdx=%d curIdx=%d\n", dstIdx, curIdx);

        const int2 dstNodeStartAccept =
            getStartAndAccept(graphDP1GPU, graphDP2GPU, dstNodePair);

        // We track if any two arcs outgoing from this node pair match
        // on epsilon. We record if they do.
        if (graphDP1GPU.olabels[firstArcIdx] != epsilon) {
          generateCombinedGraphNodesAndArcs(
              dstIdx,
              curIdx,
              make_int2(firstArcIdx, secondArcIdx),
              dstNodeStartAccept,
              reachableGPU,
              newNodesOffsetGPU,
              newNodesVisitedGPU,
              toExploreGPU,
              gradInfoFirstGPU,
              gradInfoSecondGPU,
              newGraphDPGPU,
              graphDP1GPU.ilabels[firstArcIdx],
              graphDP2GPU.olabels[secondArcIdx],
              graphDP1GPU.weights[firstArcIdx] + graphDP2GPU.weights[secondArcIdx]);
        }
      }

      // The epsilon matches
      if (result.checkEpsilonArcPair.x &&
          (!epsilonMatch || graphDP2GPU.accept[result.nodePair.y] ||
           !graphDP1GPU.accept[result.nodePair.x]) &&
          (graphDP1GPU.olabels[firstArcIdx] == epsilon)) {
        // When arc from first node has epsilon label then we consider
        // second node
        int2 dstNodePair = make_int2(
            graphDP1GPU.dstNodes[firstArcIdx], result.nodePair.y);
        const int dstIdx = TwoDToOneDIndex(
            dstNodePair.x, dstNodePair.y, numNodesFirst);
        const int curIdx = TwoDToOneDIndex(
            result.nodePair.x, result.nodePair.y, numNodesFirst);

	// printf("krn2b dstIdx=%d curIdx=%d\n", dstIdx, curIdx);

        const int2 dstNodeStartAccept =
            getStartAndAccept(graphDP1GPU, graphDP2GPU, dstNodePair);

        generateCombinedGraphNodesAndArcs(
            dstIdx,
            curIdx,
            make_int2(firstArcIdx, -1),
            dstNodeStartAccept,
            reachableGPU,
            newNodesOffsetGPU,
            newNodesVisitedGPU,
            toExploreGPU,
            gradInfoFirstGPU,
            gradInfoSecondGPU,
            newGraphDPGPU,
            graphDP1GPU.ilabels[firstArcIdx],
            epsilon,
            graphDP1GPU.weights[firstArcIdx]);
      }

      // The epsilon matches
      if (result.checkEpsilonArcPair.y &&
          (!epsilonMatch || graphDP1GPU.accept[result.nodePair.x]) &&
          (graphDP2GPU.ilabels[secondArcIdx] == epsilon)) {
        // When arc from second node has epsilon label then we consider
        // first node
        int2 dstNodePair = make_int2(
            result.nodePair.x, graphDP2GPU.dstNodes[secondArcIdx]);
        const int dstIdx = TwoDToOneDIndex(
            dstNodePair.x, dstNodePair.y, numNodesFirst);
        const int curIdx = TwoDToOneDIndex(
            result.nodePair.x, result.nodePair.y, numNodesFirst);

	// printf("krn2c dstIdx=%d curIdx=%d\n", dstIdx, curIdx);
	
        const int2 dstNodeStartAndAccept =
            getStartAndAccept(graphDP1GPU, graphDP2GPU, dstNodePair);

        generateCombinedGraphNodesAndArcs(
            dstIdx,
            curIdx,
            make_int2(-1, secondArcIdx),
            dstNodeStartAndAccept,
            reachableGPU,
            newNodesOffsetGPU,
            newNodesVisitedGPU,
            toExploreGPU,
            gradInfoFirstGPU,
            gradInfoSecondGPU,
            newGraphDPGPU,
            epsilon,
            graphDP2GPU.olabels[secondArcIdx],
            graphDP2GPU.weights[secondArcIdx]);
      }
    }
  }
}

} // namespace

Graph compose(const Graph& first, const Graph& second) {
  GraphDataParallel graphDP1, graphDP2;

  // Convert from AOS to SOA
  graphDP1 = convertToDataParallel(first);
  graphDP2 = convertToDataParallel(second);

  // Copy to GPU
  GraphDataParallelGPU graphDP1GPU, graphDP2GPU;
  graphDP1GPU = copyToGPU(graphDP1);
  graphDP2GPU = copyToGPU(graphDP2);
  
  const int numAllPairNodes = first.numNodes() * second.numNodes();
  const int numNodesFirst = first.numNodes();

  // Fixed number of CUDA threads and stream for all kernels
  const int NT = 128;

  //////////////////////////////////////////////////////////////////////////
  // Step 1: Data parallel findReachable
  //////////////////////////////////////////////////////////////////////////
  std::vector<int> reachable(numAllPairNodes, false);
  std::vector<int> epsilonMatched(numAllPairNodes, false);

  std::vector<int> toExplore(numAllPairNodes, false);

  int* reachableGPU;
  int* epsilonMatchedGPU;
  int* toExploreGPU;

  hipMalloc((void **)(&reachableGPU), sizeof(int) * numAllPairNodes);
  hipMalloc((void **)(&epsilonMatchedGPU), sizeof(int) * numAllPairNodes);
  hipMalloc((void **)(&toExploreGPU), sizeof(int) * numAllPairNodes);

  hipMemcpy((void *)epsilonMatchedGPU, (void *)(epsilonMatched.data()), sizeof(int) * numAllPairNodes, hipMemcpyHostToDevice);

  {
    std::vector<int> acceptDP1 = convertToNodes(graphDP1.accept);
    std::vector<int> acceptDP2 = convertToNodes(graphDP2.accept);

    for (auto f : acceptDP1) {
      for (auto s : acceptDP2) {
        toExplore[TwoDToOneDIndex(f, s, numNodesFirst)] = true;
        reachable[TwoDToOneDIndex(f, s, numNodesFirst)] = true;
      }
    }
  }

  // std::cout << "num all pair nodes " << numAllPairNodes << std::endl;
  hipMemcpy(reachableGPU, (void *)(reachable.data()), sizeof(int) * numAllPairNodes, hipMemcpyHostToDevice);
  hipMemcpy((void *)toExploreGPU, (void *)(toExplore.data()), sizeof(int) * numAllPairNodes, hipMemcpyHostToDevice);

  // This is the outer control loop that would spawn DP kernels
  while(checkAnyTrueGPU(toExploreGPU, numAllPairNodes)) {
    // Convert bits set in toExplore to node pairs
    hipMemcpy((void *)(toExplore.data()), (void *)(toExploreGPU), sizeof(int) * numAllPairNodes, hipMemcpyDeviceToHost);
    auto toExploreNodePair = convertToNodePair(toExplore, numNodesFirst);
    assert(toExploreNodePair.first.size() == toExploreNodePair.second.size());

    int* toExploreNodePairFirstGPU;
    int* toExploreNodePairSecondGPU;
    hipMalloc((void **)(&toExploreNodePairFirstGPU), sizeof(int) * toExploreNodePair.first.size());
    hipMalloc((void **)(&toExploreNodePairSecondGPU), sizeof(int) * toExploreNodePair.second.size());
    hipMemcpy((void *)toExploreNodePairFirstGPU, (void *)(toExploreNodePair.first.data()),
		    sizeof(int) * toExploreNodePair.first.size(), hipMemcpyHostToDevice);
    hipMemcpy((void *)toExploreNodePairSecondGPU, (void *)(toExploreNodePair.second.data()),
		    sizeof(int) * toExploreNodePair.second.size(), hipMemcpyHostToDevice);

    // Reset so pristine state for next frontier to explore
    hipMemset((void*)toExploreGPU, false, sizeof(int) * numAllPairNodes);

    const size_t numToExploreNodePair = toExploreNodePair.first.size();

    int* tVecGPU;
    int* toExploreNumArcsFirstGPU;
    int* toExploreNumArcsSecondGPU;

    std::tie(tVecGPU, toExploreNumArcsFirstGPU, toExploreNumArcsSecondGPU) =
      calculateArcCrossProductOffsetGPU(toExploreNodePairFirstGPU, toExploreNodePairSecondGPU,
        numToExploreNodePair, graphDP1GPU, graphDP2GPU, true);

    int* arcCrossProductOffsetGPU;
    size_t numArcCrossProductOffset;
    int totalArcs;
    {
      std::tie(arcCrossProductOffsetGPU, numArcCrossProductOffset, totalArcs) = prefixSumScanGPU(tVecGPU, numToExploreNodePair, true);
      assert(numArcCrossProductOffset == (numToExploreNodePair + 1));

      hipFree(tVecGPU);
    }

    const int gridSize = div_up(totalArcs, NT);

    findReachableKernel<<<gridSize, NT, 0, 0>>>(graphDP1GPU, graphDP2GPU, arcCrossProductOffsetGPU,
		    toExploreNumArcsFirstGPU, toExploreNumArcsSecondGPU, toExploreNodePairFirstGPU,
		    toExploreNodePairSecondGPU, numNodesFirst, totalArcs, numArcCrossProductOffset,
		    toExploreGPU, reachableGPU, epsilonMatchedGPU);

    hipFree(toExploreNodePairFirstGPU);
    hipFree(toExploreNodePairSecondGPU);
    hipFree(arcCrossProductOffsetGPU);
    hipFree(toExploreNumArcsFirstGPU);
    hipFree(toExploreNumArcsSecondGPU);
  } // end while for findReachable

  // Copy back to CPU
  hipMemcpy((void *)(reachable.data()), reachableGPU, sizeof(int) * numAllPairNodes, hipMemcpyDeviceToHost);

  //////////////////////////////////////////////////////////////////////////
  // Step 2: Compute a) valid nodes in combined graph
  //                 b) Number of in and out arcs in combined graph
  // This information is used to generate offsets for nodes and arcs
  // in the combined graph
  //////////////////////////////////////////////////////////////////////////
  std::vector<int> newNodes(numAllPairNodes, false);

  // Number of in and out arcs per node
  std::vector<int> numOutArcs(numAllPairNodes, 0);
  std::vector<int> numInArcs(numAllPairNodes, 0);

  int* newNodesGPU;
  int* numOutArcsGPU;
  int* numInArcsGPU;

  hipMalloc((void **)(&newNodesGPU), sizeof(int) * numAllPairNodes);
  hipMalloc((void **)(&numOutArcsGPU), sizeof(int) * numAllPairNodes);
  hipMalloc((void **)(&numInArcsGPU), sizeof(int) * numAllPairNodes);

  // Tracks the nodes that are going to be present in the combined graph
  std::fill(toExplore.begin(), toExplore.end(), false);

  {
    std::vector<int> startDP1 = convertToNodes(graphDP1.start);
    std::vector<int> startDP2 = convertToNodes(graphDP2.start);

    for (auto f : startDP1) {
      for (auto s : startDP2) {
        auto startIdx = TwoDToOneDIndex(f, s, numNodesFirst);
        if (reachable[startIdx]) {
          toExplore[startIdx] = true;
          newNodes[startIdx] = true;
        }
      }
    }
  }

  hipMemcpy(newNodesGPU, (void *)(newNodes.data()), sizeof(int) * numAllPairNodes, hipMemcpyHostToDevice);
  hipMemcpy(numOutArcsGPU, (void *)(numOutArcs.data()), sizeof(int) * numAllPairNodes, hipMemcpyHostToDevice);
  hipMemcpy(numInArcsGPU, (void *)(numInArcs.data()), sizeof(int) * numAllPairNodes, hipMemcpyHostToDevice);

  // This is the outer control loop that would spawn DP kernels
  while (checkAnyTrue(toExplore)) {
    // Convert bits set in toExplore to node pairs
    auto toExploreNodePair = convertToNodePair(toExplore, numNodesFirst);

    std::vector<int> arcCrossProductOffset;
    std::pair<std::vector<int>, std::vector<int>> toExploreNumArcs;
    std::tie(arcCrossProductOffset, toExploreNumArcs) =
        calculateArcCrossProductOffset(
            toExploreNodePair, graphDP1, graphDP2, false);

    const int totalArcs = prefixSumScan(arcCrossProductOffset, true);
    const int gridSize = div_up(totalArcs, NT);

    // std::cout << "phase 2 totalArcs " << totalArcs << std::endl;
    // std::cout << "phase 2 gridSize " << gridSize << std::endl;
 
    // Reset to pristine state for next frontier to explore
    std::fill(toExplore.begin(), toExplore.end(), false);
    hipMemcpy((void *)toExploreGPU, (void *)(toExplore.data()), sizeof(int) * numAllPairNodes, hipMemcpyHostToDevice);

    if (totalArcs > 0) {
      int* toExploreNodePairFirstGPU;
      int* toExploreNodePairSecondGPU;
      hipMalloc((void **)(&toExploreNodePairFirstGPU), sizeof(int) * toExploreNodePair.first.size());
      hipMalloc((void **)(&toExploreNodePairSecondGPU), sizeof(int) * toExploreNodePair.second.size());
      hipMemcpy((void *)toExploreNodePairFirstGPU, (void *)(toExploreNodePair.first.data()),
		      sizeof(int) * toExploreNodePair.first.size(), hipMemcpyHostToDevice);
      hipMemcpy((void *)toExploreNodePairSecondGPU, (void *)(toExploreNodePair.second.data()),
		      sizeof(int) * toExploreNodePair.second.size(), hipMemcpyHostToDevice);

      int* arcCrossProductOffsetGPU;
      hipMalloc((void **)(&arcCrossProductOffsetGPU), sizeof(int) * arcCrossProductOffset.size());
      hipMemcpy((void *)arcCrossProductOffsetGPU, (void *)(arcCrossProductOffset.data()),
		      sizeof(int) * arcCrossProductOffset.size(), hipMemcpyHostToDevice);

      int* toExploreNumArcsFirstGPU;
      int* toExploreNumArcsSecondGPU;
      hipMalloc((void **)(&toExploreNumArcsFirstGPU), sizeof(int) * toExploreNumArcs.first.size());
      hipMalloc((void **)(&toExploreNumArcsSecondGPU), sizeof(int) * toExploreNumArcs.second.size());
      hipMemcpy((void *)toExploreNumArcsFirstGPU, (void *)(toExploreNumArcs.first.data()),
		      sizeof(int) * toExploreNumArcs.first.size(), hipMemcpyHostToDevice);
      hipMemcpy((void *)toExploreNumArcsSecondGPU, (void *)(toExploreNumArcs.second.data()),
		      sizeof(int) * toExploreNumArcs.second.size(), hipMemcpyHostToDevice);

      computeValidNodeAndArcKernel<<<gridSize, NT, 0, 0>>>(graphDP1GPU, graphDP2GPU,
        arcCrossProductOffsetGPU, toExploreNumArcsFirstGPU, toExploreNumArcsSecondGPU,
        toExploreNodePairFirstGPU, toExploreNodePairSecondGPU, reachableGPU,
        epsilonMatchedGPU, numNodesFirst, totalArcs, arcCrossProductOffset.size(),
        toExploreGPU, newNodesGPU, numInArcsGPU, numOutArcsGPU);

      hipMemcpy((void *)(toExplore.data()), (void *)(toExploreGPU), sizeof(int) * numAllPairNodes, hipMemcpyDeviceToHost);

      hipFree(toExploreNodePairFirstGPU);
      hipFree(toExploreNodePairSecondGPU);
      hipFree(arcCrossProductOffsetGPU);
      hipFree(toExploreNumArcsFirstGPU);
      hipFree(toExploreNumArcsSecondGPU);
    }
  }

  //////////////////////////////////////////////////////////////////////////
  // Step 3: Generate offsets for nodes and arcs in combined graph
  //////////////////////////////////////////////////////////////////////////
  // Copy back generated data to CPU
  hipMemcpy((void *)(newNodes.data()), (void *)(newNodesGPU), sizeof(int) * numAllPairNodes, hipMemcpyDeviceToHost);
  hipMemcpy((void *)(numInArcs.data()), (void *)(numInArcsGPU), sizeof(int) * numAllPairNodes, hipMemcpyDeviceToHost);
  hipMemcpy((void *)(numOutArcs.data()), (void *)(numOutArcsGPU), sizeof(int) * numAllPairNodes, hipMemcpyDeviceToHost);

  // Generate offsets for nodes and arcs
  GraphDataParallel newGraphDP;

  // Convert bool array to int for prefix sum
  // Record arc offsets for new nodes in new graph
  std::vector<int> newNodesOffset(newNodes.size(), 0);
  for (size_t i = 0; i < newNodes.size(); ++i) {
    if (newNodes[i]) {
      // std::cout << "d " << i << " " << numInArcs[i] << " " << numOutArcs[i] << std::endl;
      newNodesOffset[i] = 1;
      newGraphDP.inArcOffset.push_back(numInArcs[i]);
      newGraphDP.outArcOffset.push_back(numOutArcs[i]);
    }
  }

  const int totalNodes = prefixSumScan(newNodesOffset, false);

  // Check that number of nodes match
  assert(totalNodes == newGraphDP.inArcOffset.size());
  assert(newGraphDP.inArcOffset.size() == newGraphDP.outArcOffset.size());

  // Prefix sum to generate offsets
  const int totalInArcs = prefixSumScan(newGraphDP.inArcOffset, false);
  const int totalOutArcs = prefixSumScan(newGraphDP.outArcOffset, false);

  // Allocate space for start and accept nodes
  assert(newGraphDP.start.empty());
  assert(newGraphDP.accept.empty());
  newGraphDP.start.resize(totalNodes, false);
  newGraphDP.accept.resize(totalNodes, false);

  // This is the total number of arcs and they must be equal
  assert(totalInArcs == totalOutArcs);

  newGraphDP.inArcs.resize(totalInArcs);
  newGraphDP.outArcs.resize(totalOutArcs);
  newGraphDP.ilabels.resize(totalOutArcs);
  newGraphDP.olabels.resize(totalOutArcs);
  newGraphDP.srcNodes.resize(totalOutArcs);
  newGraphDP.dstNodes.resize(totalOutArcs);
  newGraphDP.weights.resize(totalOutArcs);

  // std::cout << "totalInArcs " << totalInArcs << " totalOutArcs " << totalOutArcs << std::endl;

  // SOA for gradInfo
  std::pair<std::vector<int>, std::vector<int>> gradInfo;
  gradInfo.first.resize(totalOutArcs);
  gradInfo.second.resize(totalOutArcs);

  int *gradInfoFirstGPU;
  int *gradInfoSecondGPU;

  hipMalloc((void **)(&gradInfoFirstGPU), sizeof(int) * totalOutArcs);
  hipMalloc((void **)(&gradInfoSecondGPU), sizeof(int) * totalOutArcs);

  //////////////////////////////////////////////////////////////////////////
  // Step 4: Generate nodes and arcs in combined graph
  //////////////////////////////////////////////////////////////////////////
  std::fill(toExplore.begin(), toExplore.end(), false);
  std::vector<int> newNodesVisited(numAllPairNodes, false);

  {
    std::vector<int> startDP1 = convertToNodes(graphDP1.start);
    std::vector<int> startDP2 = convertToNodes(graphDP2.start);

    for (auto f : startDP1) {
      for (auto s : startDP2) {
        const int nodeIdx = TwoDToOneDIndex(f, s, numNodesFirst);
        if (reachable[nodeIdx]) {
          toExplore[nodeIdx] = true;
          newNodesVisited[nodeIdx] = true;
          newGraphDP.start[newNodesOffset[nodeIdx]] = true;
          newGraphDP.accept[newNodesOffset[nodeIdx]] =
              graphDP1.accept[f] && graphDP2.accept[s];
        }
      }
    }
  }

  GraphDataParallelGPU newGraphDPGPU;
  newGraphDPGPU = copyToGPU(newGraphDP);

  int* newNodesVisitedGPU;
  hipMalloc((void **)(&newNodesVisitedGPU), sizeof(int) * numAllPairNodes);
  hipMemcpy((void *)newNodesVisitedGPU, (void *)(newNodesVisited.data()), sizeof(int) * numAllPairNodes, hipMemcpyHostToDevice);

  int* newNodesOffsetGPU;
  hipMalloc((void **)(&newNodesOffsetGPU), sizeof(int) * numAllPairNodes);
  hipMemcpy((void *)newNodesOffsetGPU, (void *)(newNodesOffset.data()), sizeof(int) * numAllPairNodes, hipMemcpyHostToDevice);

  // This is the outer control loop that would spawn DP kernels
  while (checkAnyTrue(toExplore)) {
    // Convert bits set in toExplore to node pairs
    auto toExploreNodePair = convertToNodePair(toExplore, numNodesFirst);
    // std::cout << "pass" << std::endl;
    /*
    for (auto i : toExploreNodePair.first) {
       std::cout << "f " << i << std::endl;
    }

    std::cout << "====" << std::endl;
    for (auto i : toExploreNodePair.second) {
       std::cout << "f " << i << std::endl;
    }*/

    std::vector<int> arcCrossProductOffset;
    std::pair<std::vector<int>, std::vector<int>> toExploreNumArcs;
    std::tie(arcCrossProductOffset, toExploreNumArcs) =
        calculateArcCrossProductOffset(
            toExploreNodePair, graphDP1, graphDP2, false);

    const int totalArcs = prefixSumScan(arcCrossProductOffset, true);
    const int gridSize = div_up(totalArcs, NT);

    // std::cout << "totalArcs " << totalArcs << std::endl;
    // Reset so pristine state for next frontier to explore
    // No dependence between iterations
    std::fill(toExplore.begin(), toExplore.end(), false);
    hipMemcpy((void *)toExploreGPU, (void *)(toExplore.data()), sizeof(int) * numAllPairNodes, hipMemcpyHostToDevice);

    if (totalArcs > 0) {
      int* toExploreNodePairFirstGPU;
      int* toExploreNodePairSecondGPU;
      hipMalloc((void **)(&toExploreNodePairFirstGPU), sizeof(int) * toExploreNodePair.first.size());
      hipMalloc((void **)(&toExploreNodePairSecondGPU), sizeof(int) * toExploreNodePair.second.size());
      hipMemcpy((void *)toExploreNodePairFirstGPU, (void *)(toExploreNodePair.first.data()),
		      sizeof(int) * toExploreNodePair.first.size(), hipMemcpyHostToDevice);
      hipMemcpy((void *)toExploreNodePairSecondGPU, (void *)(toExploreNodePair.second.data()),
		      sizeof(int) * toExploreNodePair.second.size(), hipMemcpyHostToDevice);

      int* arcCrossProductOffsetGPU;
      hipMalloc((void **)(&arcCrossProductOffsetGPU), sizeof(int) * arcCrossProductOffset.size());
      hipMemcpy((void *)arcCrossProductOffsetGPU, (void *)(arcCrossProductOffset.data()),
		      sizeof(int) * arcCrossProductOffset.size(), hipMemcpyHostToDevice);

      int* toExploreNumArcsFirstGPU;
      int* toExploreNumArcsSecondGPU;
      hipMalloc((void **)(&toExploreNumArcsFirstGPU), sizeof(int) * toExploreNumArcs.first.size());
      hipMalloc((void **)(&toExploreNumArcsSecondGPU), sizeof(int) * toExploreNumArcs.second.size());
      hipMemcpy((void *)toExploreNumArcsFirstGPU, (void *)(toExploreNumArcs.first.data()),
		      sizeof(int) * toExploreNumArcs.first.size(), hipMemcpyHostToDevice);
      hipMemcpy((void *)toExploreNumArcsSecondGPU, (void *)(toExploreNumArcs.second.data()),
		      sizeof(int) * toExploreNumArcs.second.size(), hipMemcpyHostToDevice);

      generateNodeAndArcKernel<<<gridSize, NT, 0, 0>>>(graphDP1GPU, graphDP2GPU,
        arcCrossProductOffsetGPU, toExploreNumArcsFirstGPU, toExploreNumArcsSecondGPU,
        toExploreNodePairFirstGPU, toExploreNodePairSecondGPU, reachableGPU,
        epsilonMatchedGPU, numNodesFirst, totalArcs, arcCrossProductOffset.size(),
        newGraphDPGPU, toExploreGPU, gradInfoFirstGPU, gradInfoSecondGPU,
        newNodesOffsetGPU, newNodesVisitedGPU);

      hipMemcpy((void *)(toExplore.data()), (void *)(toExploreGPU), sizeof(int) * numAllPairNodes, hipMemcpyDeviceToHost);

      hipFree(toExploreNodePairFirstGPU);
      hipFree(toExploreNodePairSecondGPU);
      hipFree(arcCrossProductOffsetGPU);
      hipFree(toExploreNumArcsFirstGPU);
      hipFree(toExploreNumArcsSecondGPU);
    }
  }

  // Copy graph on GPU to CPU
  hipMemcpy((void *)(newGraphDP.accept.data()), (void *)(newGraphDPGPU.accept), sizeof(int) * newGraphDPGPU.numNodes, hipMemcpyDeviceToHost);
  hipMemcpy((void *)(newGraphDP.start.data()), (void *)(newGraphDPGPU.start), sizeof(int) * newGraphDPGPU.numNodes, hipMemcpyDeviceToHost);
  hipMemcpy((void *)(newGraphDP.inArcs.data()), (void *)(newGraphDPGPU.inArcs), sizeof(int) * newGraphDPGPU.numArcs, hipMemcpyDeviceToHost);
  hipMemcpy((void *)(newGraphDP.outArcs.data()), (void *)(newGraphDPGPU.outArcs), sizeof(int) * newGraphDPGPU.numArcs, hipMemcpyDeviceToHost);
  hipMemcpy((void *)(newGraphDP.ilabels.data()), (void *)(newGraphDPGPU.ilabels), sizeof(int) * newGraphDPGPU.numArcs, hipMemcpyDeviceToHost);
  hipMemcpy((void *)(newGraphDP.olabels.data()), (void *)(newGraphDPGPU.olabels), sizeof(int) * newGraphDPGPU.numArcs, hipMemcpyDeviceToHost);
  hipMemcpy((void *)(newGraphDP.srcNodes.data()), (void *)(newGraphDPGPU.srcNodes), sizeof(int) * newGraphDPGPU.numArcs, hipMemcpyDeviceToHost);
  hipMemcpy((void *)(newGraphDP.dstNodes.data()), (void *)(newGraphDPGPU.dstNodes), sizeof(int) * newGraphDPGPU.numArcs, hipMemcpyDeviceToHost);
  hipMemcpy((void *)(newGraphDP.weights.data()), (void *)(newGraphDPGPU.weights), sizeof(float) * newGraphDPGPU.numArcs, hipMemcpyDeviceToHost);

  assert(newGraphDPGPU.numArcs == totalOutArcs);
  hipMemcpy((void *)(gradInfo.first.data()), (void *)(gradInfoFirstGPU), sizeof(int) * totalOutArcs, hipMemcpyDeviceToHost);
  hipMemcpy((void *)(gradInfo.second.data()), (void *)(gradInfoSecondGPU), sizeof(int) * totalOutArcs, hipMemcpyDeviceToHost);

  if (0)
  {
    std::cout << "nodes " << newGraphDP.inArcOffset.size() << std::endl;
    std::cout << "nodes " << newGraphDP.outArcOffset.size() << std::endl;

    std::cout << "inArcOffset" << std::endl;
    for (auto i : newGraphDP.inArcOffset) {
      std::cout << i << std::endl;
    }

    std::cout << "outArcOffset" << std::endl;
    for (auto i : newGraphDP.outArcOffset) {
      std::cout << i << std::endl;
    }

    std::cout << "inArcs" << std::endl;
    for (auto i : newGraphDP.inArcs) {
      std::cout << i << std::endl;
    }

    std::cout << "outArcs" << std::endl;
    for (auto i : newGraphDP.outArcs) {
      std::cout << i << std::endl;
    }

    std::cout << "ilabels" << std::endl;
    for (auto i : newGraphDP.ilabels) {
      std::cout << i << std::endl;
    }

    std::cout << "olabels" << std::endl;
    for (auto i : newGraphDP.olabels) {
      std::cout << i << std::endl;
    }

    std::cout << "srcNodes" << std::endl;
    for (auto i : newGraphDP.srcNodes) {
      std::cout << i << std::endl;
    }

    std::cout << "dstNodes" << std::endl;
    for (auto i : newGraphDP.dstNodes) {
      std::cout << i << std::endl;
    }

    std::cout << "weights" << std::endl;
    for (auto i : newGraphDP.weights) {
      std::cout << i << std::endl;
    }
  }
  // Not needed since the CPU data is never incremented
  // Shift offset values back down after adding arcs to newGraphDP
  // The offset values got converted from exclusive prefix sum to inclusive
  // Need to convert them back to exclusive prefix sum  by starting with 0
  // and shifting to right by 1
  // for (int i = newGraphDP.outArcOffset.size() - 1; i >= 0; --i) {
    // newGraphDP.outArcOffset[i] = i == 0 ? 0 : newGraphDP.outArcOffset[i - 1];
    // newGraphDP.inArcOffset[i] = i == 0 ? 0 : newGraphDP.inArcOffset[i - 1];
  // }

  // Convert back and add in autograd metadata
  auto nGraph = convertFromDataParallel(newGraphDP);
  nGraph.setInputs({first, second});

  if (0)
  {
    std::cout << "numNodes " << nGraph.numNodes() << std::endl;

    std::cout << "accept" << std::endl;
    for (auto i : nGraph.accept()) {
      std::cout << i << std::endl;
    }

    std::cout << "start" << std::endl;
    for (auto i : nGraph.start()) {
      std::cout << i << std::endl;
    }

    std::cout << "numIn" << std::endl;
    for (int i = 0; i < nGraph.numNodes(); ++i) {
      std::cout << nGraph.numIn(i) << std::endl;
    }

    std::cout << "numOut" << std::endl;
    for (int i = 0; i < nGraph.numNodes(); ++i) {
      std::cout << nGraph.numOut(i) << std::endl;
    }
  }

  // Convert gradInfo SOA to AOS
  std::vector<std::pair<int, int>> gradInfoAOS;
  for (int i = 0; i < gradInfo.first.size(); ++i) {
    gradInfoAOS.emplace_back(gradInfo.first[i], gradInfo.second[i]);
  }

  // TODO eliminate this copy pasta.
  auto gradFunc = [gradInfo = std::move(gradInfoAOS)](
                      std::vector<Graph>& inputs, Graph deltas) {
    // In this case the arc's parents are always from the
    // first and second input graphs respectively.
    bool calcGrad1 = inputs[0].calcGrad();
    bool calcGrad2 = inputs[1].calcGrad();
    auto grad1 = calcGrad1 ? std::vector<float>(inputs[0].numArcs(), 0.0)
                           : std::vector<float>{};
    auto grad2 = calcGrad2 ? std::vector<float>(inputs[1].numArcs(), 0.0)
                           : std::vector<float>{};
    for (int i = 0; i < gradInfo.size(); i++) {
      auto arcGrad = deltas.weight(i);
      auto& arcs = gradInfo[i];
      if (calcGrad1 && arcs.first >= 0) {
        grad1[arcs.first] += arcGrad;
      }
      if (calcGrad2 && arcs.second >= 0) {
        grad2[arcs.second] += arcGrad;
      }
    }
    inputs[0].addGrad(std::move(grad1));
    inputs[1].addGrad(std::move(grad2));
  };
  nGraph.setGradFunc(std::move(gradFunc));
  return nGraph;
}

} // namespace dataparallel
} // namespace detail
} // namespace gtn

    /*
    if (0)
    {
      int *aCPGPU;
      int *tEN1GPU;
      int *tEN2GPU;

      std::tie(aCPGPU, tEN1GPU, tEN2GPU) = calculateArcCrossProductOffsetGPU(
        toExploreNodePairFirstGPU, toExploreNodePairSecondGPU,
        toExploreNodePair.first.size(), graphDP1GPU, graphDP2GPU, true);

      std::vector<int> aCP(numToExploreNodePair);
      std::vector<int> tEN1(numToExploreNodePair);
      std::vector<int> tEN2(numToExploreNodePair);

      hipMemcpy((void *)(aCP.data()), (void *)(aCPGPU), sizeof(int) * numToExploreNodePair, hipMemcpyDeviceToHost);
      hipMemcpy((void *)(tEN1.data()), (void *)(tEN1GPU), sizeof(int) * numToExploreNodePair, hipMemcpyDeviceToHost);
      hipMemcpy((void *)(tEN2.data()), (void *)(tEN2GPU), sizeof(int) * numToExploreNodePair, hipMemcpyDeviceToHost);

      assert(std::equal(arcCrossProductOffset.begin(), arcCrossProductOffset.end(), aCP.begin()));
      assert(std::equal(toExploreNumArcs.first.begin(), toExploreNumArcs.first.end(), tEN1.begin()));
      assert(std::equal(toExploreNumArcs.second.begin(), toExploreNumArcs.second.end(), tEN2.begin()));

      hipFree(aCPGPU);
      hipFree(tEN1GPU);
      hipFree(tEN2GPU);
    }*/

    /*
    if(0)
    {
      std::vector<int> tVec(arcCrossProductOffset);
      const size_t numElts = tVec.size();
      int* tVecGPU;
      hipMalloc((void **)(&tVecGPU), sizeof(int) * numElts);
      hipMemcpy((void *)tVecGPU, (void *)(tVec.data()), sizeof(int) * numElts, hipMemcpyHostToDevice);

      const int totalArcs = prefixSumScan(tVec, true);
      int* tVecScanGPU;
      size_t tVecScanElts;
      int tArcsGPU;
      std::tie(tVecScanGPU, tVecScanElts, tArcsGPU) = prefixSumScanGPU(tVecGPU, numElts, true);

      assert(tVec.size() == (numElts + 1));
      assert(tVecScanElts == (numElts + 1));
      std::vector<int> tVecNew(tVec.size());
      hipMemcpy((void *)(tVecNew.data()), (void *)(tVecScanGPU), sizeof(int) * tVecScanElts, hipMemcpyDeviceToHost);

      assert(totalArcs == tArcsGPU);
      assert(std::equal(tVec.begin(), tVec.end(), tVecNew.begin()));

      hipFree(tVecGPU);
      hipFree(tVecScanGPU);
    }*/

